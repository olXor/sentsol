#include "hip/hip_runtime.h"
#include "thoughtkernel.cuh"

__host__ __device__ float thoughtTransferFunction(float in) {
#if THOUGHT_TRANSFER == RECTIFIER
	if (in / TRANSFER_WIDTH > TRANSFER_FUNCTION_LIMIT)
		return in;
	if (in / TRANSFER_WIDTH < -TRANSFER_FUNCTION_LIMIT)
		return NEGATIVE_TRANSFER_FACTOR*in;
	return TRANSFER_WIDTH*(log(1.0f + exp(in / TRANSFER_WIDTH)) - NEGATIVE_TRANSFER_FACTOR*log(1.0f + exp(-in / TRANSFER_WIDTH)));
#elif THOUGHT_TRANSFER == SIGMOID
	if (in / TRANSFER_WIDTH > TRANSFER_FUNCTION_LIMIT)
		return 1.0f;
	if (in / TRANSFER_WIDTH < -TRANSFER_FUNCTION_LIMIT)
		return 0.0f;
	return 1.0f / (1.0f + exp(-in / TRANSFER_WIDTH));
#else
	return 0.0f;
#endif
}

__host__ __device__ float thoughtTransferDerivative(float in) {
#if THOUGHT_TRANSFER == RECTIFIER
	if (in / TRANSFER_WIDTH > TRANSFER_FUNCTION_LIMIT)
		return 1.0f;
	if (in / TRANSFER_WIDTH < -TRANSFER_FUNCTION_LIMIT)
		return NEGATIVE_TRANSFER_FACTOR;
	return 1.0f / (1.0f + exp(-in / TRANSFER_WIDTH)) + NEGATIVE_TRANSFER_FACTOR / (1.0f + exp(in / TRANSFER_WIDTH));
#elif THOUGHT_TRANSFER == SIGMOID
	float tf = thoughtTransferFunction(in);
	return tf*(1-tf);
#else
	return 0.0f;
#endif
}

#ifdef MAX_WEIGHT_CHANGE
__device__ float thoughtBoundChange(float change) {
	if (change > MAX_WEIGHT_CHANGE)
		change = MAX_WEIGHT_CHANGE;
	else if (change < -MAX_WEIGHT_CHANGE)
		change = -MAX_WEIGHT_CHANGE;
	return change;
}
#endif

__device__ bool thoughtIsNan(float num) {
	return !isfinite(num);
}

__device__ void thoughtSumVector(float* vec, size_t size, size_t threadNum, size_t numThreads) {
	size_t stride = 1;
	while (stride < size) {
		for (size_t j = 2 * stride*threadNum; j + stride < size; j += 2 * stride*numThreads) {
			vec[j] += vec[j + stride];
		}
		stride *= 2;
		__syncthreads();
	}
}

__global__ void computeThoughtLayer(ThoughtMatrices* tm, ThoughtParameters* tp, size_t bpTurn, size_t prevTurn) {
	size_t outNeuron = blockIdx.x;
	size_t clusterStart = outNeuron - outNeuron%CLUSTER_SIZE;
	size_t inConnection = threadIdx.x;
	size_t numInThreads = tp->forBlockX;

	size_t backCon = tp->backwardConnectivity;
	size_t sideCon = tp->sideConnectivity;
	size_t totalCon = backCon + sideCon;

	size_t numInputs = tp->numInputs;
	size_t numOutputs = tp->numOutputs;

	float* inlayer = &tm->inlayer[prevTurn*numInputs];
	float* outlayer = &tm->outlayer[bpTurn*numOutputs];
	float* prevoutlayer = &tm->outlayer[prevTurn*numOutputs];
	float* outTDs = &tm->outTDs[bpTurn*numOutputs];

	extern __shared__ float outputs[];

	if (bpTurn != prevTurn) {
		for (size_t i = inConnection; i < totalCon; i += numInThreads) {
			if (i < backCon)
				outputs[i] = tm->weights[i + totalCon*outNeuron] * inlayer[(clusterStart + i) % numInputs];
			else
				outputs[i] = tm->weights[i + totalCon*outNeuron] * prevoutlayer[(clusterStart + i - backCon) % numOutputs];
		}

		__syncthreads();

		thoughtSumVector(outputs, totalCon, inConnection, numInThreads);
	}
	else {
		outputs[0] = 0;
	}

	__shared__ float randFact;
	if (threadIdx.x == 0) {
		randFact = THOUGHT_RAND_WIDTH * hiprand_normal(&tm->randStates[outNeuron]);
	}
	
	__syncthreads();

	if (threadIdx.x == 0) {
		outlayer[outNeuron] = thoughtTransferFunction(outputs[0] - tm->thresholds[outNeuron] + randFact);
	}
	else if (threadIdx.x == 1 % numInThreads) {
		outTDs[outNeuron] = thoughtTransferDerivative(outputs[0] - tm->thresholds[outNeuron] + randFact);
	}
}

size_t getThoughtComputeSharedSize(ThoughtParameters* tp) {
	size_t size = 0;
	size += tp->backwardConnectivity + tp->sideConnectivity;
	size *= sizeof(float);
	return size;
}

__global__ void backPropagateThoughtLayer(ThoughtMatrices* tm, ThoughtParameters* tp, size_t bpTurn) {
	size_t outNeuron = blockIdx.x;
	size_t clusterStart = outNeuron - outNeuron%CLUSTER_SIZE;
	size_t inConnection = threadIdx.x;
	size_t numInThreads = tp->backBlockX;

	size_t backCon = tp->backwardConnectivity;
	size_t sideCon = tp->sideConnectivity;
	size_t totalCon = backCon + sideCon;

	size_t numInputs = tp->numInputs;
	size_t numOutputs = tp->numOutputs;
	
	float* inlayer = &tm->inlayer[bpTurn*numInputs];
	float* prevoutlayer = &tm->outlayer[(bpTurn-1)*numOutputs];
	float* outTDs = &tm->outTDs[bpTurn*numOutputs];
	float* errors = &tm->errors[bpTurn*numOutputs];
	float* inerrors = tm->inerrors;
	if (inerrors != NULL)
		inerrors = &inerrors[bpTurn*numInputs];
	float* preverrors = &tm->errors[(bpTurn - 1)*numOutputs];

	float outErrorTD = errors[outNeuron] * outTDs[outNeuron];

	if (inConnection == 0) {
#ifdef MAX_WEIGHT_CHANGE
		float change = thoughtBoundChange(outErrorTD);
#else
		float change = outErrorTD;
#endif
		tm->thresholds[outNeuron] += change;
	}

	for (size_t i = inConnection; i < totalCon; i += numInThreads) {
		float change;
		size_t weightNum = i + totalCon*outNeuron;
		float weight = tm->weights[weightNum];
		if (i < backCon) {
			size_t inNeuron = (clusterStart + i) % numInputs;
			change = outErrorTD * inlayer[inNeuron];
			if (inerrors != NULL)
				inerrors[inNeuron] += outErrorTD * weight;
		}
		else {
			size_t prevNeuron = (clusterStart + i - backCon) % numOutputs;
			change = outErrorTD * prevoutlayer[prevNeuron];
			preverrors[prevNeuron] += outErrorTD * weight;
		}
#ifdef MAX_WEIGHT_CHANGE
		change = thoughtBoundChange(change);
#endif
		tm->weights[weightNum] = weight - change;
	}
}

size_t getThoughtBackPropSharedSize(ThoughtParameters* tp) {
	return 0;
}

__global__ void copyThoughtKernelOutputToHost(ThoughtMatrices* tm, ThoughtParameters* tp, float* hostoutput, size_t bpTurn) {
	size_t outNeuron = threadIdx.x;
	float* currentoutput = &tm->outlayer[bpTurn*tp->numOutputs];
	hostoutput[outNeuron] = currentoutput[outNeuron];
}

//if we expand to > 1024 neurons per layer we'll have to break this up into blocks
__global__ void initRandomStates(ThoughtMatrices* tm, ThoughtParameters* tp, size_t seed, size_t sequenceStart) {
	size_t outNeuron = threadIdx.x;
	size_t seq = sequenceStart + outNeuron;
	hiprand_init(seed, seq, 0, &tm->randStates[outNeuron]);
}