#include "hip/hip_runtime.h"
#include "thoughtnet.cuh"

ThoughtNet::ThoughtNet(size_t nInputs, size_t nOutputs, size_t nLayers, size_t nClusters) {
	numInputs = nInputs;
	numOutputs = nOutputs;
	numLayers = nLayers;
	numClusters = nClusters;
	thoughtCollection = createThoughtCollection();
}

ThoughtCollection ThoughtNet::createThoughtCollection() {
	ThoughtCollection tc;

	tc.numThoughtLayers = numLayers;
	tc.thoughtMats.resize(numLayers);
	tc.thoughtPars.resize(numLayers);

	//set up ThoughtParameters
	for (size_t i = 0; i < numLayers; i++) {
		ThoughtParameters* tp = &tc.thoughtPars[i];
		if (i == 0)
			tp->numInputs = numInputs;
		else
			tp->numInputs = numClusters*CLUSTER_SIZE;

		if (i == numLayers - 1)
			tp->numOutputs = numOutputs;
		else
			tp->numOutputs = numClusters*CLUSTER_SIZE;

		tp->backwardConnectivity = std::min(2 * CLUSTER_SIZE, (int)tp->numInputs);
		tp->sideConnectivity = std::min(CLUSTER_SIZE, (int)tp->numOutputs);

		//---thread shaping
		tp->forNBlockX = tp->numOutputs;
		tp->forBlockX = tp->backwardConnectivity + tp->sideConnectivity;

		tp->backNBlockX = tp->numOutputs;
		tp->backBlockX = tp->backwardConnectivity + tp->sideConnectivity;
		//-----------------

		instantiateThoughtMatrices(&tc.thoughtMats[i], &tc.thoughtPars[i]);
	}
	linkThoughtLayers(&tc);
	copyThoughtLayersToDevice(&tc);
}

ThoughtCollection ThoughtNet::getThoughtCollection() {
	return thoughtCollection;
}

ThoughtMatrices instantiateThoughtMatrices(ThoughtMatrices* tm, ThoughtParameters* tp) {
	checkCudaErrors(hipMalloc(&tm->inlayer1, tp->numInputs*sizeof(float)));
	checkCudaErrors(hipMalloc(&tm->inlayer2, tp->numInputs*sizeof(float)));
	checkCudaErrors(hipMalloc(&tm->outlayer1, tp->numOutputs*sizeof(float)));
	checkCudaErrors(hipMalloc(&tm->outlayer2, tp->numOutputs*sizeof(float)));

	size_t totalConnectivity = tp->backwardConnectivity + tp->sideConnectivity;
	size_t numWeights = tp->numOutputs*totalConnectivity;

	float* h_weights = new float[numWeights];
	for (size_t i = 0; i < numWeights; i++) {
		h_weights[i] = (rand() % 21 - 10.0f) / 10.0f / (totalConnectivity + 1);
	}
	checkCudaErrors(hipMalloc(&tm->weights, numWeights*sizeof(float)));
	checkCudaErrors(hipMemcpy(tm->weights, h_weights, numWeights*sizeof(float), hipMemcpyHostToDevice));
	delete [] h_weights;

	size_t numThresholds = tp->numOutputs;
	float* h_thresholds = new float[numThresholds];
	for (size_t i = 0; i < numThresholds; i++) {
		h_thresholds[i] = (rand() % 21 - 10.0f) / 10.0f / (totalConnectivity + 1);
	}
	checkCudaErrors(hipMalloc(&tm->thresholds, numThresholds*sizeof(float)));
	checkCudaErrors(hipMemcpy(tm->thresholds, h_thresholds, numThresholds*sizeof(float), hipMemcpyHostToDevice));
	delete [] h_thresholds;

	checkCudaErrors(hipMalloc(&tm->outTDs, tp->numOutputs*sizeof(float)));
	checkCudaErrors(hipMalloc(&tm->errors, tp->numOutputs*sizeof(float)));

	tm->forwardSharedMem = getThoughtComputeSharedSize(tp);
	tm->backwardSharedMem = getThoughtBackPropSharedSize(tp);
}

void linkThoughtLayers(ThoughtCollection* tc) {
	for (size_t i = 1; i < tc->numThoughtLayers; i++) {
		ThoughtMatrices* mat = &tc->thoughtMats[i];
		ThoughtMatrices* prevMat = &tc->thoughtMats[i-1];
		ThoughtParameters* par = &tc->thoughtPars[i];
		ThoughtParameters* prevPar = &tc->thoughtPars[i-1];

		if (prevPar->numOutputs != par->numInputs) {
			std::cout << "Layer sizes didn't match during link step" << std::endl;
			throw new std::runtime_error("Layer sizes didn't match during link step");
		}

		checkCudaErrors(hipFree(mat->inlayer1));
		mat->inlayer1 = prevMat->outlayer1;
		checkCudaErrors(hipFree(mat->inlayer2));
		mat->inlayer2 = prevMat->outlayer2;
	}
}

void copyThoughtLayersToDevice(ThoughtCollection* tc) {
	for (size_t i = 0; i < tc->thoughtMats.size(); i++){
		ThoughtMatrices* d_thoughtMat;
		checkCudaErrors(hipMalloc(&d_thoughtMat, sizeof(ThoughtMatrices)));
		checkCudaErrors(hipMemcpy(d_thoughtMat, &tc->thoughtMats[i], sizeof(ThoughtMatrices), hipMemcpyHostToDevice));
		tc->d_thoughtMats.push_back(d_thoughtMat);
	}

	for (size_t i = 0; i < tc->thoughtPars.size(); i++){
		ThoughtParameters* d_thoughtPar;
		checkCudaErrors(hipMalloc(&d_thoughtPar, sizeof(ThoughtParameters)));
		checkCudaErrors(hipMemcpy(d_thoughtPar, &tc->thoughtPars[i], sizeof(ThoughtParameters), hipMemcpyHostToDevice));
		tc->d_thoughtPars.push_back(d_thoughtPar);
	}
}

ThoughtNet::~ThoughtNet() {
	for (size_t i = 0; i < thoughtCollection.numThoughtLayers; i++) {
		ThoughtMatrices tm = thoughtCollection.thoughtMats[i];
		checkCudaErrors(hipFree(tm.inlayer1));
		checkCudaErrors(hipFree(tm.inlayer2));
		checkCudaErrors(hipFree(tm.outlayer1));
		checkCudaErrors(hipFree(tm.outlayer2));
		checkCudaErrors(hipFree(tm.weights));
		checkCudaErrors(hipFree(tm.thresholds));
		checkCudaErrors(hipFree(tm.outTDs));
		checkCudaErrors(hipFree(tm.errors));

		checkCudaErrors(hipFree(thoughtCollection.d_thoughtMats[i]));
		checkCudaErrors(hipFree(thoughtCollection.d_thoughtPars[i]));
	}
}

void ThoughtNet::incrementTurn() {
	turn++;
}

bool ThoughtNet::turn1Front() {
	return turn % 2 == 0;
}

size_t ThoughtNet::getNumInputs() {
	return numInputs;
}

size_t ThoughtNet::getNumOutputs() {
	return numOutputs;
}

size_t ThoughtNet::getNumLayers() {
	return numLayers;
}

size_t ThoughtNet::getNumClusters() {
	return numClusters;
}

void ThoughtNet::compute() {
	bool turn1front = turn1Front();
	for (size_t i = 0; i < thoughtCollection.numThoughtLayers; i++) {
		ThoughtMatrices* tm = &thoughtCollection.thoughtMats[i];
		ThoughtParameters* tp = &thoughtCollection.thoughtPars[i];
		dim3 nBlocks(tp->forNBlockX);
		dim3 shape(tp->forBlockX);
		size_t shared = tm->forwardSharedMem;
		computeThoughtLayer<<<nBlocks, shape, shared>>>(tm, tp, turn1front);
		checkCudaErrors(hipPeekAtLastError());
	}
}

//note that the order of execution shouldn't matter here
void ThoughtNet::backPropagate() {
	bool turn1front = turn1Front();
	for (size_t i = 0; i < thoughtCollection.numThoughtLayers; i++) {
		ThoughtMatrices* tm = &thoughtCollection.thoughtMats[i];
		ThoughtParameters* tp = &thoughtCollection.thoughtPars[i];
		dim3 nBlocks(tp->backNBlockX);
		dim3 shape(tp->backBlockX);
		size_t shared = tm->backwardSharedMem;
		backPropagateThoughtLayer<<<nBlocks, shape, shared>>>(tm, tp, turn1front);
		checkCudaErrors(hipPeekAtLastError());
	}
}

//note: returns the BACK input layer, so that one can immediately call calculate afterwards
float* ThoughtNet::getDeviceInputLayer() {
	bool turn1front = turn1Front();
	if (turn1front)
		return thoughtCollection.thoughtMats[0].inlayer2;
	else
		return thoughtCollection.thoughtMats[0].inlayer1;
}

//note: returns the FRONT output layer, so that one can immediately call this after calculating
float* ThoughtNet::getDeviceOutputLayer() {
	bool turn1front = turn1Front();
	if (turn1front)
		return thoughtCollection.thoughtMats[thoughtCollection.numThoughtLayers - 1].outlayer1;
	else
		return thoughtCollection.thoughtMats[thoughtCollection.numThoughtLayers - 1].outlayer2;
}

ThoughtMatrices* ThoughtNet::getLastLevelMatrices() {
	return &thoughtCollection.thoughtMats[numLayers - 1];
}

ThoughtParameters* ThoughtNet::getLastLevelParameters() {
	return &thoughtCollection.thoughtPars[numLayers - 1];
}