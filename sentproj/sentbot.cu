#include "sentbot.cuh"
#include <sstream>

SentBot::SentBot(size_t nInputs, size_t nOutputs, size_t nLayers, size_t nClusters) {
	thoughtNet = new ThoughtNet(nInputs, nOutputs, nLayers, nClusters);
	valueNet = new ValueNet(thoughtNet);
	numInputs = nInputs;
	numOutputs = nOutputs;
	h_inputs = new float[numInputs];
	checkCudaErrors(hipHostAlloc(&h_outputs, numOutputs*sizeof(float), hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer(&d_outputs, h_outputs, 0));
}

SentBot::~SentBot() {
	delete thoughtNet;
	delete valueNet;
	delete[] h_inputs;
	delete[] h_outputs;
}

void SentBot::takeTurn() {
	thoughtNet->incrementTurn();
	valueNet->incrementTurn();
	float* d_inputs = thoughtNet->getDeviceInputLayer();
	//use the below if we're not using mapped memory for the output
	//float* d_outputs = thoughtNet->getDeviceOutputLayer();

	hipEvent_t calcDone;
	checkCudaErrors(hipEventCreate(&calcDone));

	checkCudaErrors(hipMemcpyAsync(d_inputs, h_inputs, thoughtNet->getNumInputs()*sizeof(float), hipMemcpyHostToDevice));

	thoughtNet->compute();

	thoughtNet->copyOutputToHost(d_outputs);
	checkCudaErrors(hipPeekAtLastError());

	checkCudaErrors(hipEventRecord(calcDone));

	valueNet->compute();

	valueNet->backPropagate();

	thoughtNet->backPropagate();

	checkCudaErrors(hipEventSynchronize(calcDone));
}

void SentBot::givePleasurePain(float pleasurePain) {
	valueNet->updateWeights(pleasurePain);
}

void SentBot::saveWeights(std::string fname) {
	std::stringstream base;
	base << "saveweights/" << fname;

	std::stringstream tss;
	tss << base.str() << "thought";
	thoughtNet->saveWeights(tss.str());

	std::stringstream vss;
	vss << base.str() << "value";
	valueNet->saveWeights(vss.str());
}