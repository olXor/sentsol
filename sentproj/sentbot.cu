#include "sentbot.cuh"

SentBot::SentBot(size_t nInputs, size_t nOutputs, size_t nLayers, size_t nClusters) {
	thoughtNet = new ThoughtNet(nInputs, nOutputs, nLayers, nClusters);
	valueNet = new ValueNet(thoughtNet);
	numInputs = nInputs;
	numOutputs = nOutputs;
	h_inputs = new float[numInputs];
	checkCudaErrors(hipHostAlloc(&h_outputs, numOutputs*sizeof(float), hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer(&d_outputs, h_outputs, 0));
}

SentBot::~SentBot() {
	delete thoughtNet;
	delete valueNet;
	delete[] h_inputs;
	delete[] h_outputs;
}

void SentBot::takeTurn() {
	thoughtNet->incrementTurn();
	valueNet->incrementTurn();
	float* d_inputs = thoughtNet->getDeviceInputLayer();
	//use the below if we're not using mapped memory for the output
	//float* d_outputs = thoughtNet->getDeviceOutputLayer();

	hipEvent_t calcDone;
	checkCudaErrors(hipEventCreate(&calcDone));

	checkCudaErrors(hipMemcpyAsync(d_inputs, h_inputs, thoughtNet->getNumInputs()*sizeof(float), hipMemcpyHostToDevice));

	thoughtNet->compute();

	copyOutputToHost << <1, 1, 0 >> >(thoughtNet->getLastLevelMatrices(), thoughtNet->getLastLevelParameters(), d_outputs, thoughtNet->turn1Front());
	checkCudaErrors(hipPeekAtLastError());

	checkCudaErrors(hipEventRecord(calcDone));

	valueNet->compute();

	valueNet->backPropagate();

	thoughtNet->backPropagate();

	checkCudaErrors(hipEventSynchronize(calcDone));
}

void SentBot::givePleasurePain(float pleasurePain) {
	valueNet->updateWeights(pleasurePain);
}